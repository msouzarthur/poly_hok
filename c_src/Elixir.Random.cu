
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__device__ float atomic_random() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state);
    return hiprand_uniform(&state);
}