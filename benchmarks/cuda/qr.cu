#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void dot_product_kernel(float* A, float* Q, float* R, int k, int n) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if (i <= k || i >= n) return;

    float dot = 0.0f;
    for (int j=0; j<n; j++) {
        dot += Q[j*n+k]*A[j*n+i];
    }
    R[k*n+i] = dot;
}

__global__ void update_column_kernel(float* A, float* Q, float* R, int k, int n) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int col = k+1+i;
    if (col >= n) return;

    for (int row=0; row<n; row++) {
        A[row*n+col] -= R[k*n+col]*Q[row*n+k];
    }
}

__global__ void normalize_column_kernel(float* A, float* Q, float* R, int k, int n) {
    __shared__ float sum[256];
    int tid = threadIdx.x;
    int idx = blockIdx.x*blockDim.x+tid;

    float val = 0;
    if (idx < n) {
        val = A[idx*n+k];
        sum[tid] = val*val;
    } else {
        sum[tid] = 0.0f;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) sum[tid] += sum[tid + stride];
        __syncthreads();
    }

    float norm = sqrtf(sum[0]);
    if (norm < 1e-10f) norm = 1.0f; // para evitar divisão por zero

    if (idx < n) {
        Q[idx*n+k] = A[idx*n+k] / norm;
    }
    if (tid == 0) {
        R[k*n+k] = norm;
    }
}

int main(int argc, char *argv[]) {
    hipEvent_t start, stop;   
    hipError_t nb_error;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);  

    int n_samples = atoi(argv[1]);
    
    float time;
    float *A_host;
    float *A_dev;
    float *Q_dev;
    float *R_dev;

    A_host = (float *)malloc(n_samples*n_samples*sizeof(float));

    for (int i=0; i<n_samples*n_samples; i++) {
        A_host[i] = (float)(rand()%100)/100.0f;
    }

    hipMalloc(&A_dev, n_samples*n_samples*sizeof(float));
    hipMalloc(&Q_dev, n_samples*n_samples*sizeof(float));
    hipMalloc(&R_dev, n_samples*n_samples*sizeof(float));

    hipMemcpy(A_dev, A_host, n_samples*n_samples*sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n_samples+threads-1)/threads;

    for (int k=0; k<n_samples; k++) {
        normalize_column_kernel<<<blocks, threads>>>(A_dev, Q_dev, R_dev, k, n_samples);
        hipDeviceSynchronize();

        dot_product_kernel<<<blocks, threads>>>(A_dev, Q_dev, R_dev, k, n_samples);
        hipDeviceSynchronize();

        update_column_kernel<<<blocks, threads>>>(A_dev, Q_dev, R_dev, k, n_samples);
        hipDeviceSynchronize();
    }

    //
    hipMemcpy(A_host, Q_dev, n_samples*n_samples*sizeof(float), hipMemcpyDeviceToHost);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    // 
    printf("Q[0,0] = %f\n", A_host[0]);

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;
    //
    hipFree(A_dev);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    //
    hipFree(Q_dev);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    //
    hipFree(R_dev);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    //
    hipEventDestroy(start);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 6: %s\n", hipGetErrorString(nb_error));
    //
    hipEventDestroy(stop);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(nb_error));
    //
    free(A_host);

    return 0;
}
