#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(float* d_t_matrix, int n_states) {
    
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= n_states) return;

    hiprandState state;
    hiprand_init(clock64() + idx, 0, 0, &state);

    float sum = 0.0f;

    for (int next_state = 0; next_state < n_states; ++next_state) {
        float value = hiprand_uniform(&state);
        d_t_matrix[idx*n_states + next_state] = value;
        sum += value;
    }

    for (int next_state = 0; next_state < n_states; ++next_state) {
        d_t_matrix[idx*n_states + next_state] /= sum;
    }

}

__global__ void markov(float* d_t_matrix, float* d_state_vector, float* d_next_state_vector, int n_states) {
    
    int state = blockIdx.x*blockDim.x + threadIdx.x;
    if (state >= n_states) return;

    float sum = 0.0f;
    for (int prev_state = 0; prev_state < n_states; ++prev_state) {
        sum += d_t_matrix[prev_state*n_states + state]*d_state_vector[prev_state];
    }
    d_next_state_vector[state] = sum;
    
}

int main(int argc, char** argv) {
    hipError_t nb_error;
    hipEvent_t start, stop;   

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);  

    int n_states = atoi(argv[1]);
    int n_steps = atoi(argv[2]);
    int s_blocks = 128;
    int n_blocks = (n_states + s_blocks - 1) / s_blocks;

    float *d_t_matrix, *d_state_vector, *d_next_state_vector;
    float time;

    // 
    hipMalloc(&d_t_matrix, n_states*n_states*sizeof(float));
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(nb_error));
    // 
    // 
    hipMalloc(&d_state_vector, n_states*sizeof(float));
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(nb_error));
    // 
    // 
    hipMalloc(&d_next_state_vector, n_states*sizeof(float));
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    // 

    setup_kernel<<<n_blocks, s_blocks>>>(d_t_matrix, n_states);
    hipDeviceSynchronize();

    float* h_vector = (float*)calloc(n_states, sizeof(float));
    h_vector[0] = 1.0f;
    hipMemcpy(d_state_vector, h_vector, n_states*sizeof(float), hipMemcpyHostToDevice);
    free(h_vector);

    for (int i = 0; i < n_steps; ++i) {
        markov<<<n_blocks, s_blocks>>>(d_t_matrix, d_state_vector, d_next_state_vector, n_states);
        hipDeviceSynchronize();

        float* tmp = d_state_vector;
        d_state_vector = d_next_state_vector;
        d_next_state_vector = tmp;
    }

    float* resultadoFinal = (float*)malloc(n_states*sizeof(float));
    hipMemcpy(resultadoFinal, d_state_vector, n_states*sizeof(float), hipMemcpyDeviceToHost);
    printf("Resultado final:\n");
    for (int i = 0; i < n_states; ++i) {
        printf("%f ", resultadoFinal[i]);
    }
    printf("\n");
    
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("tempo: %3.1fms\n", time);
    
    hipFree(d_t_matrix);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));

    hipFree(d_state_vector);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));

    hipFree(d_next_state_vector);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    
    hipEventDestroy(start);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 6: %s\n", hipGetErrorString(nb_error));
    
    hipEventDestroy(stop);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(nb_error));
    
    free(resultadoFinal);

    return 0;
}
