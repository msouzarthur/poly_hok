#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void cholesky_diag(float *A, int k, int n) {
    __shared__ float sum[256];
    int tid = threadIdx.x;
    float temp = 0;

    for (int j=tid; j<k; j+=blockDim.x) {
        float val = A[k*n+j];
        temp += val*val;
    }
    sum[tid] = temp;
    __syncthreads();

    for (int stride=blockDim.x/2; stride>0; stride/=2) {
        if (tid<stride) sum[tid] += sum[tid+stride];
        __syncthreads();
    }

    if (tid==0) {
        A[k*n+k] = sqrtf(A[k*n+k] - sum[0]);
    }
}

__global__ void cholesky_update_col(float *A, int k, int n) {
    int i = blockIdx.x*blockDim.x+threadIdx.x+k+1;
    if (i<n) {
        float temp = 0;
        for (int j=0; j<k; j++) {
            temp += A[i*n+j]*A[k*n+j];
        }
        A[i*n+k] = (A[i*n+k]-temp)/A[k*n+k];
    }
}

int main(int argc, char *argv[]) {
    hipEvent_t start, stop;   
    hipError_t nb_error;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int n_samples = atoi(argv[1]);
    int s_blocks = 128;
    // int n_blocks = (n_samples+s_blocks-1)/s_blocks;

    float time;
    float *A_host;
    float *A_dev;
    
    A_host = (float *)malloc(n_samples*n_samples*sizeof(float));

    for (int i=0; i<n_samples; i++) {
        for (int j=0; j<n_samples; j++) {
            if (i==j)
                A_host[i*n_samples+j] = 4.0f;
            else
                A_host[i*n_samples+j] = 1.0f;
        }
    }

    for (int i=0; i<n_samples; i++) {
        for (int j=0; j<n_samples; j++) {
            printf("%f\t", A_host[i*n_samples+j]);
        }
        printf("\n");
    }
    //
    hipMalloc(&A_dev, n_samples*n_samples*sizeof(float));
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(nb_error));
    //
    hipMemcpy(A_dev, A_host, n_samples * n_samples * sizeof(float), hipMemcpyHostToDevice);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(nb_error));
    //

    for (int k = 0; k < n_samples; k++) {
        cholesky_diag<<<1, s_blocks>>>(A_dev, k, n_samples);
        hipDeviceSynchronize();

        int remaining = n_samples - (k + 1);
        int n_blocks_update = (remaining + s_blocks - 1) / s_blocks;
        if (n_blocks_update > 0) {
            cholesky_update_col<<<n_blocks_update, s_blocks>>>(A_dev, k, n_samples);
            hipDeviceSynchronize();
        }
    }

    //
    hipMemcpy(A_host, A_dev, n_samples*n_samples*sizeof(float), hipMemcpyDeviceToHost);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(nb_error));
    //

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;
    
    printf("resultado:\n");

    for (int i = 0; i < n_samples; i++) {
        for (int j = 0; j < n_samples; j++) {
            printf("%f\t", A_host[i*n_samples+j]);  
        }
        printf("\n");
    }
    //
    hipFree(A_dev);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    //
    hipEventDestroy(start);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    //
    hipEventDestroy(stop);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    //

    free(A_host);
    
    return 0;
}
