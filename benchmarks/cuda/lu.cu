
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void gen_matrix(float* A, int r) {
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    if (idx >= r*r) return;

    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);

    A[idx] = hiprand_uniform(&state) * 10.0f;
}

// Kernel para um passo da decomposição LU
__global__ void lu_kernel(float* A, float* L, float* U, int r, int i) {
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    if (idx >= r) return;

    // Diagonal de L
    if (idx == i) {
        L[i*r+i] = 1.0f;
    }

    // Linha i de U
    if (idx >= i) {
        float soma = 0.0f;
        for (int j=0; j<i; j++) {
            soma += L[i*r+j]*U[j*r+idx];
        }
        U[i*r+idx] = A[i*r+idx]-soma;
    }

    // Coluna i de L
    if (idx > i) {
        float soma = 0.0f;
        for (int j=0; j<i; j++) {
            soma += L[idx*r+j]*U[j*r+i];
        }
        L[idx*r+i] = (A[idx*r+i]-soma)/U[i*r+i];
    }
}

void print_matrix(float* data, int r) {
    for (int i=0; i<r; i++) {
        for (int j=0; j<r; j++) {
            printf("%8.4f ", data[i*r+j]);
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {
    hipError_t nb_error;
    hipEvent_t start, stop;   

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int r = atoi(argv[1]);
    size_t tamanho = r*r*sizeof(float);

    float *d_matrix, *d_l, *d_u;
    float *A, *L, *U;
    float time;

    // 
    hipMalloc(&d_matrix, tamanho);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(nb_error));
    // 
    //
    hipMalloc(&d_l, tamanho);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(nb_error));
    // 
    // 
    hipMalloc(&d_u, tamanho);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    // 
    // Inicialização da matriz A com números aleatórios
    gen_matrix<<<(r*r+255)/256, 256>>>(d_matrix, r);
    hipDeviceSynchronize();

    // Decomposição LU paralela (passo a passo)
    for (int i = 0; i < r; i++) {
        lu_kernel<<<(r+255)/256, 256>>>(d_matrix, d_l, d_u, r, i);
        hipDeviceSynchronize();
    }
    
    // Alocação no host e cópia dos resultados
    A = (float*)malloc(tamanho);
    L = (float*)malloc(tamanho);
    U = (float*)malloc(tamanho);

    hipMemcpy(A, d_matrix, tamanho, hipMemcpyDeviceToHost);
    hipMemcpy(L, d_l, tamanho, hipMemcpyDeviceToHost);
    hipMemcpy(U, d_u, tamanho, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;
    
    printf("tempo: %3.1fms\n", time);
    // printf("matriz A (original):\n");
    // print_matrix(A, r);
    // printf("matriz L (inferior):\n");
    // print_matrix(L, r);
    // printf("matriz U (superior):\n");
    // print_matrix(U, r);

    // Libera memória
    free(A);
    free(L);
    free(U);

    hipFree(d_matrix); 
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));

    hipFree(d_l); 
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 5: %s\n", hipGetErrorString(nb_error));

    hipFree(d_u);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 6: %s\n", hipGetErrorString(nb_error));

    hipEventDestroy(start);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(nb_error));

    hipEventDestroy(stop);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 8: %s\n", hipGetErrorString(nb_error));

    return 0;
}
